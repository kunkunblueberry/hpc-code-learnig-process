#include "hip/hip_runtime.h"
//��һ���汾���е�ѭ��չ���Ѿ�ӵ�кܲ���������ˣ����������취Ҳֻ�����Ƚ����޵���������


/*
����˼��
�ڹ�Լ��ģ�ﵽһ��warp֮ǰ��������Ҫͬ�����������ܹ���ȷ��ȡ
��������֪��һ��warp�����в�����һ���ģ�Ҳ����simt������������һ������ͬ����
����֮���ﵽһ��warp�Ĺ�ģ�£�����Ҫͬ������ʱ��
*/
//Ҫ�ӹؼ���volatile
//volatile �Ǹ������ӵ�һ�� �������ǡ����������þ�һ�仰�����߱����� �����������ֵ���ܻᱻ�㿴�����Ĵ���͵͵�ġ���
// ���Ա�Ϲ�Ż���ÿ����������ȥshared�ڴ����������Ҳ������д���ڴ档
//������������������Ż�if�����´𰸴���

#include<iostream>
#include "hip/hip_runtime.h"

#include ""
#include <stdio.h>

// ���CUDA�����Ƿ�ɹ�
#define CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "Error: %s in file %s, line %d\n", \
                    hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define THREAD_PER_BLOCK 256

__global__ void reduce(float* d_input, float* d_output) {
	float* input_begin = d_input +2* blockDim.x * blockIdx.x;
	volatile __shared__ float shared[THREAD_PER_BLOCK];
	shared[threadIdx.x] = input_begin[threadIdx.x]+input_begin[threadIdx.x+blockDim.x];
	__syncthreads();

	//for (int i=blockDim.x/2; i > 32; i /= 2) {		
	//	if (threadIdx.x <i) {
	//		shared[threadIdx.x] += shared[threadIdx.x + i];
	//	}
	//	__syncthreads();
	//}
	//����˵���ǰ����ѭ��һ��һ��д����������forѭ���Ŀ���

	if (threadIdx.x < THREAD_PER_BLOCK / 2)//128
	{
		shared[threadIdx.x] += shared[threadIdx.x + blockDim.x / 2];
	}
	__syncthreads();
	if (threadIdx.x < THREAD_PER_BLOCK / 4)//64
	{
		shared[threadIdx.x] += shared[threadIdx.x + blockDim.x / 4];
	}
	__syncthreads();

	//�����ȱ���ǲ��÷�װ����װ̫���ѣ�THREAD_PER_BLOCK���˺󣬴�������ҲҪ��


	if (threadIdx.x < 32) {
		shared[threadIdx.x] += shared[threadIdx.x + 32];
		shared[threadIdx.x] += shared[threadIdx.x + 16];
		shared[threadIdx.x] += shared[threadIdx.x + 8];
		shared[threadIdx.x] += shared[threadIdx.x + 4];
		shared[threadIdx.x] += shared[threadIdx.x + 2];
		shared[threadIdx.x] += shared[threadIdx.x + 1];
	}

	if (threadIdx.x == 0)
		d_output[blockIdx.x] = shared[0];
}

int main() {
	const int N = 32 * 1024 * 1024;
	float* input = (float*)malloc(N * sizeof(float));
	float* d_input;
	CHECK(hipMalloc((void**)&d_input, N * sizeof(float)));

	int block_num = N / THREAD_PER_BLOCK/2;		//������һ��shared���ص�ʱ���������block�����������߳̿����
	float* output = (float*)malloc(block_num * sizeof(float));
	float* d_output;
	CHECK(hipMalloc((void**)&d_output, block_num * sizeof(float)));		//���ﶼ����Ϊblock_num��������һ��

	float* result = (float*)malloc(block_num * sizeof(float));	//��Ϊ�����ǽ�������Է���block_num*sizeof�������������N*sizeof

	for (int i = 0; i < N; i++) {
		input[i] = 2.0 * (float)rand() - 1.0;
	}

	//cpu����Ҫ���ģ�block�Ĳ����������룬�̹߳���ӱ�
	for (int i = 0; i < block_num; i++) {
		float cur = 0;
		for (int j = 0; j < 2*THREAD_PER_BLOCK; j++) {
			cur += input[i *2* THREAD_PER_BLOCK + j];
		}
		result[i] = cur;
	}

	CHECK(hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice));

	dim3 grid(block_num, 1);
	dim3 block(THREAD_PER_BLOCK, 1);

	reduce << <grid, block >> > (d_input, d_output);

	float* out = (float*)malloc(block_num * sizeof(float));
	CHECK(hipMemcpy(out, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost));
	for (int i = 0; i < block_num; i++) {
		if (abs(out[i] - result[i]) > 0.0005) {
			printf("����ȣ�����");
			break;
		}
	}
	printf("���");
	hipFree(d_input);
	hipFree(d_output);
	free(input);
	free(output);
	free(result);
	free(out);
	return 0;
}

/*
�� CUDA �У������ֺ��ķ�ʽ����ʵ��ѭ��չ����Loop Unrolling������ һ���Ǳ������Զ��Ż���
��һ����ͨ��#pragma unroll����ָ���ֶ����ơ�ѭ��չ���ı����� ����ѭ�����еĵ�������ֱ��չ��Ϊ�������롱��
�Ӷ�����ѭ�����ƣ��������жϡ��������������Ŀ�����ͬʱΪ�������ṩ����ָ����У�ILP�����Ż��ռ䣬�����ʺϹ�Լ������˷��ȸ�Ƶѭ������

#pragma unroll�� CUDA������ C/C++ ��չ���ṩ�ı���ָ�������ʽ����ѭ����չ����Ϊ�����ȼ����ڱ��������Զ��Ż�����������ʹ����ʽ�����ǲ�ͬ����

ָ����ʽ	����˵��
#pragma unroll			�ñ����� ���Զ�����չ����������ͨ����չ�����е�����ǰ����ѭ�������Ǳ����ڳ�����
#pragma unroll N		ǿ��չ��ΪN�ε�����N��������������ѭ���ܴ���С��N������Զ���չ����
#pragma unroll 1		ǿ�Ʋ�չ��ѭ�������ñ��������Զ�չ�������ڵ��Ի����ⳡ����

�������Զ��Ż�Ҳ����ѭ��չ����������������ֶ������ߵ��Ż��������ܽ���������������
*/
